
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#define Mask_Width 101


void Convolution1D(int *N, int *P, int *M, int n) {
    int i, j;

    for(i=0; i < n; i++){
        int Pvalue = 0;
        int N_start_point = i - ((Mask_Width-1)/2);
        for(j = 0; j < Mask_Width; j++){
            if(N_start_point+j >=0 && N_start_point+j < n){
                Pvalue += N[N_start_point+j]*M[j];
            }
        }
        P[i] = Pvalue;
    }
}

int main(int argc, char *argv[]) {

    /* input, output e máscara */ 
    int *N , *P, *M; 
    int n, i;

    /* Tamanho do vetor */	
    scanf("%d",&n);

    /* Alocação dos buffers necessários */
    P = (int *)malloc(n*sizeof(int));
    N = (int *)malloc(n*sizeof(int));
    M = (int *)malloc(sizeof(int)*Mask_Width);

    /* entrada dos valores */
    for(i = 0; i < n ; i++)
        scanf("%d",&N[i]);

    for(i = 0; i < Mask_Width; i++) M[i] = i;

    Convolution1D(N, P, M, n);

    for(i = 0; i < n; i++) printf("%d ", P[i]);
    printf("\n");

    free(P);
    free(N);
    free(M);
}
